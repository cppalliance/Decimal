#include "hip/hip_runtime.h"

//  Copyright John Maddock 2016.
//  Copyright Matt Borland 2024.
//  Use, modification and distribution are subject to the
//  Boost Software License, Version 1.0. (See accompanying file
//  LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)

#pragma nv_diag_suppress 186

#include <iostream>
#include <iomanip>
#include <vector>
#include <boost/decimal.hpp>
#include "cuda_managed_ptr.hpp"
#include "stopwatch.hpp"

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

using float_type = boost::decimal::decimal32_fast;

/**
 * CUDA Kernel Device code
 *
 */
__global__ void cuda_test(const float_type *in, int *out, int numElements)
{
    using std::cos;
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        out[i] = static_cast<int>(signbit(in[i]) + isinf(in[i]) + isnan(in[i]) + issignaling(in[i]) + isnormal(in[i]) + isfinite(in[i]));
    }
}

/**
 * Host main routine
 */
int main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    std::cout << "[Vector operation on " << numElements << " elements]" << std::endl;

    // Allocate the managed input vector A
    cuda_managed_ptr<float_type> input_vector(numElements);

    // Allocate the managed output vector C
    cuda_managed_ptr<int> output_vector(numElements);

    // Initialize the input vectors
    for (int i = 0; i < numElements; ++i)
    {
        input_vector[i] = static_cast<float_type>(rand());
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 1024;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    std::cout << "CUDA kernel launch with " << blocksPerGrid << " blocks of " << threadsPerBlock << " threads" << std::endl;

    watch w;

    cuda_test<<<blocksPerGrid, threadsPerBlock>>>(input_vector.get(), output_vector.get(), numElements);
    hipDeviceSynchronize();

    std::cout << "CUDA kernal done in: " << w.elapsed() << "s" << std::endl;

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        std::cerr << "Failed to launch vectorAdd kernel (error code " << hipGetErrorString(err) << ")!" << std::endl;
        return EXIT_FAILURE;
    }

    // Verify that the result vector is correct
    std::vector<int> results;
    results.reserve(numElements);
    w.reset();
    for(int i = 0; i < numElements; ++i)
    {
       results.push_back(static_cast<int>(signbit(input_vector[i]) + isinf(input_vector[i]) + isnan(input_vector[i]) + issignaling(input_vector[i]) + isnormal(input_vector[i]) + isfinite(input_vector[i])));
    }
    double t = w.elapsed();
    // check the results
    for(int i = 0; i < numElements; ++i)
    {
        if (output_vector[i] != results[i])
        {
            std::cerr << "Result verification failed at element " << i << "!\n"
                      << "Cuda: " << output_vector[i] << '\n'
                      << "Serial: " << results[i] << std::endl;
            return EXIT_FAILURE;
        }
    }

    std::cout << "Test PASSED, normal calculation time: " << t << "s" << std::endl;
    std::cout << "Done\n";

    return 0;
}
