#include "hip/hip_runtime.h"

//  Copyright John Maddock 2016.
//  Copyright Matt Borland 2024.
//  Use, modification and distribution are subject to the
//  Boost Software License, Version 1.0. (See accompanying file
//  LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)

#pragma nv_diag_suppress 186

#include <iostream>
#include <iomanip>
#include <vector>
#include <random>
#include <boost/decimal.hpp>
#include "cuda_managed_ptr.hpp"
#include "stopwatch.hpp"

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

using float_type = boost::decimal::decimal32_fast;

/**
 * CUDA Kernel Device code
 *
 */
__global__ void cuda_test(const float_type *in, float_type *out, int numElements)
{
    using std::cos;
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        out[i] = in[i] * in[i];
    }
}

/**
 * Host main routine
 */
int main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    std::cout << "[Vector operation on " << numElements << " elements]" << std::endl;

    // Allocate the managed input vector A
    cuda_managed_ptr<float_type> input_vector(numElements);

    // Allocate the managed output vector C
    cuda_managed_ptr<float_type> output_vector(numElements);

    // Initialize the input vectors
    std::mt19937_64 rng(42);
    std::uniform_int_distribution<int> dist(-1000, 1000);
    for (int i = 0; i < numElements; ++i)
    {
        input_vector[i] = static_cast<float_type>(dist(rng));
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 1024;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    std::cout << "CUDA kernel launch with " << blocksPerGrid << " blocks of " << threadsPerBlock << " threads" << std::endl;

    watch w;

    cuda_test<<<blocksPerGrid, threadsPerBlock>>>(input_vector.get(), output_vector.get(), numElements);
    hipDeviceSynchronize();

    std::cout << "CUDA kernal done in: " << w.elapsed() << "s" << std::endl;

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        std::cerr << "Failed to launch vectorAdd kernel (error code " << hipGetErrorString(err) << ")!" << std::endl;
        return EXIT_FAILURE;
    }

    // Verify that the result vector is correct
    std::vector<float_type> results;
    results.reserve(numElements);
    w.reset();
    for(int i = 0; i < numElements; ++i)
    {
       results.push_back(input_vector[i] * input_vector[i]);
    }
    double t = w.elapsed();
    // check the results
    for(int i = 0; i < numElements; ++i)
    {
        if (output_vector[i] != results[i])
        {
            std::cerr << "Result verification failed at element " << i << "!\n"
                      << "Cuda: " << output_vector[i] << '\n'
                      << "Serial: " << results[i] << std::endl;
            return EXIT_FAILURE;
        }
    }

    std::cout << "Test PASSED, normal calculation time: " << t << "s" << std::endl;
    std::cout << "Done\n";

    return 0;
}
